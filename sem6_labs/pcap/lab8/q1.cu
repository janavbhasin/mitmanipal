#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"

__global__ void mataddrow(int*a,int*b,int*c,int wa)
{
    int ro=threadIdx.x;
    for(int co=0;co<wa;co++)
    {
        c[ro*wa+co]=a[ro*wa+co]+b[ro*wa+co];
    }
}
__global__ void mataddcolumn(int*a,int*b,int*c,int ha)
{
    int co=threadIdx.x,wa=blockDim.x;
    for(int ro=0;ro<ha;ro++)
    {
        c[ro*wa+co]=a[ro*wa+co]+b[ro*wa+co];
    }
}
__global__ void mataddelement(int*a,int*b,int*c)
{
    int ro=threadIdx.x,co=threadIdx.y,wa=blockDim.y;
    c[ro*wa+co]=a[ro*wa+co]+b[ro*wa+co];
}
__host__ void displayMatrix (int *mat, int h, int w) 
{
    for (int i=0; i<h; i++) 
    {
        for (int j=0; j<w; j++) 
        {
            printf("%d ", mat[i * w + j]);
        }
        printf("\n");
    }
}
int main() 
{
    int ha, wa;
    printf("Enter number of rows and columns of matrix: ");
    scanf(" %d %d", &ha,&wa);
    int *A, *B, *C;
    hipMallocManaged(&A, ha * wa * sizeof(int));
    hipMallocManaged(&B, ha * wa * sizeof(int));
    hipMallocManaged(&C, ha * wa * sizeof(int));
    printf("Enter %d elements of matrix A:\n", ha * wa);
    for (int i = 0; i < ha * wa; i++) 
    {
        scanf(" %d", &A[i]);
    }
    printf("Enter %d elements of matrix B:\n", ha * wa);
    for (int i = 0; i < ha * wa; i++)      
    {
        scanf(" %d", &B[i]);
    }
    int x = -1;
    while (1) 
    {
        printf("\n1: Row-wise \n2: Column-wise \n3: Element-wise \n0: Exit \nEnter method to use for addition: ");
        scanf(" %d", &x);
        if (x == 0)
        {
            printf("Exiting...\n");
            break;
        }
        if (x == 1)
        {
            mataddrow <<< 1, ha >>> (A, B, C, wa);
        }
        else if (x == 2)
        {
            mataddcolumn <<< 1, wa >>> (A, B, C, ha);
        }
        else 
        {
            dim3 blockDim(ha, wa);
            mataddelement <<< 1, blockDim >>> (A, B, C);
        }
        hipDeviceSynchronize(); 
        printf("\nResultant matrix C:\n");
        displayMatrix(C, ha, wa);
    }
}