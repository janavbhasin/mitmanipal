#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"

__global__ void row (int* A, int* B, int* C, int wa, int wb) 
{
    int ro = threadIdx.x, val;
    for (int co = 0; co < wb; co++) 
    {
        val = 0;
        for (int k = 0; k < wa; k++) 
        {
            val += A[ro * wa + k] * B[k * wb + co];
        }
        C[ro * wb + co] = val;
    }
}
__global__ void col (int* A, int* B, int* C, int ha, int wa) 
{
    int co = threadIdx.x, val;
    int wb = blockDim.x;
    for (int ro = 0; ro < ha; ro++) 
    {
        val = 0;
        for (int k = 0; k < wa; k++) 
        {
            val += A[ro * wa + k] * B[k * wb + co];
        }
        C[ro * wb + co] = val;
    }
}
__global__ void ele (int* A, int* B, int* C, int wa) 
{
    int ro = threadIdx.x, co = threadIdx.y;
    int wb = blockDim.y;
    int val = 0;
    for (int k = 0; k < wa; k++) 
    {
        val += A[ro * wa + k] * B[k * wb + co];
    }
    C[ro * wb + co] = val;
}
__host__ void disp (int *mat, int h, int w) 
{
    for (int i = 0; i < h; i++) 
    {
        for (int j = 0; j < w; j++)
        {
            printf("%d ", mat[i * w + j]);
        }
        printf("\n");
    }
}
int main () 
{
    int ha, wa, hb, wb;
    printf("Enter number of rows of matrix A: ");
    scanf(" %d", &ha);
    printf("Enter number of columns of matrix A: ");
    scanf(" %d", &wa);
    printf("Enter number of rows of matrix B: ");
    scanf(" %d", &hb);
    printf("Enter number of columns of matrix B: ");
    scanf(" %d", &wb);
    int *A, *B, *C;
    hipMallocManaged(&A, ha * wa * sizeof(int));
    hipMallocManaged(&B, hb * wb * sizeof(int));
    hipMallocManaged(&C, ha * wb * sizeof(int));
    printf("Enter %d elements of matrix A:\n", ha * wa);
    for (int i = 0; i < ha * wa; i++)
    {
        scanf(" %d", &A[i]);
    }
    printf("Enter %d elements of matrix B:\n", hb * wb);
    for (int i = 0; i < hb * wb; i++) 
    {
        scanf(" %d", &B[i]);
    }
    int x = -1;
    while (1) 
    {
        printf("\n1: Row-wise \n2: Column-wise \n3: Element-wise \n0: Exit \nEnter method to use for multiplication: ");
        scanf(" %d", &x);
        if (x == 0) 
        {
            printf("Exiting...\n");
            break;
        }
        if (x == 1)
        {
            row <<< 1, ha >>> (A, B, C, wa, wb);
        }
        else if (x == 2)
        {
            col <<< 1, wb >>> (A, B, C, ha, wa);
        }
        else 
        {
            dim3 blockDim(ha, wb);
            ele <<< 1, blockDim >>> (A, B, C, wa);
        }
        hipDeviceSynchronize();
        printf("\nResultant matrix C:\n");
        disp(C, ha, wb);
    }
}