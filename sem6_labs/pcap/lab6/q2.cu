#include "hip/hip_runtime.h"

#include<stdio.h>
__global__ void add_vec(int*da,int*dc,int s)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int k=0;
    for(int j=0;j<s;j++)
    {
        if((da[j]<da[i])||(da[j]==da[i]&&j>i))
        {
            k++;
        }
    }
    dc[k]=da[i];
}
int main(){
    int n;
    printf("Length of the vector : ");
    scanf("%d",&n);
    int a[n],c[n];
    int *da,*dc;
    hipMalloc((void **)&da,n*sizeof(int));
    hipMalloc((void **)&dc,n*sizeof(int));
    printf("Enter vector : ");
    for(int i=0;i<n;i++)
    {
        scanf("%d",&a[i]);
    }
    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);
    dim3 g(n,1,1);
    dim3 bl(1,1,1);
    add_vec<<<g,bl>>>(da,dc,n);
    hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<n;i++)
    {
        printf("%d\t",c[i]);
    
    }    
    printf("\n");
}