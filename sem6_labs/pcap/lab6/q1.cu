#include "hip/hip_runtime.h"

#include <stdio.h>
#define B 16
__global__ void conv(float *in, float *mask, float *out, int w, int mw) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    for (int j = 0; j < mw; j++) 
    {
        int idx = i + j - (mw - 1);
        if (idx >= 0 && idx < w) 
        {
            sum += in[idx] * mask[j];
        }
    }
    if (i < w) 
    { 
        out[i] = sum;
    }
}
void init(float *arr, int n, const char *msg) 
{
    printf("%s", msg);
    for (int i = 0; i < n; i++) 
    {
        scanf("%f", &arr[i]);
    }
}
int main() 
{
    int n, mw;
    printf("enter the size of array: ");
    scanf("%d", &n);
    printf("enter the size of mask: ");
    scanf("%d", &mw);
    float *N = (float *)malloc(n * sizeof(float));
    float *M = (float *)malloc(mw * sizeof(float));
    float *P = (float *)malloc(n * sizeof(float));
    float *d_in, *d_mask, *d_out;
    hipMalloc(&d_in, n * sizeof(float));
    hipMalloc(&d_mask, mw * sizeof(float));
    hipMalloc(&d_out, n * sizeof(float));
    init(N, n, "enter array: ");
    init(M, mw, "enter mask: ");
    hipMemcpy(d_in, N, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, M, mw * sizeof(float), hipMemcpyHostToDevice);
    dim3 a(n,1,1);
    dim3 b(1,1,1);
    conv<<<b, a>>>(d_in, d_mask, d_out, n, mw);
    hipMemcpy(P, d_out, n * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) 
    {
        printf("%f\t", P[i]);
    }
    printf("\n");
}