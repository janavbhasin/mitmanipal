#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
__global__ void calculateSine(float *a, float *sin, int n) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        sin[i] = sinf(a[i]);
    }
}
int main(void)
{
    int n;
    printf("Enter number of angles:\n");
    scanf("%d", &n);
    float*ha=(float*)malloc(sizeof(float)*n);
    float*hs=(float*)malloc(sizeof(float)*n);
    float *da,*ds;
    hipMalloc((void**)&da,n*sizeof(float));
    hipMalloc((void**)&ds,n*sizeof(float));
    printf("enter angles in radians:\n");
    for(int i=0;i<n;i++)
    {
        scanf("%f",&ha[i]);
    }
    hipMemcpy(da,ha,n*sizeof(float),hipMemcpyHostToDevice);
    int b = 256; 
    int nu =(n+b-1)/b;  
    calculateSine<<<nu,b>>>(da,ds,n);
    hipMemcpy(hs,ds,n*sizeof(float),hipMemcpyDeviceToHost);
    printf("Sine values are: \n");
    for (int i = 0; i < n; i++) 
    {
        printf("%f ", hs[i]);
    }
    printf("\n");
    free(ha);
    free(hs);
    hipFree(da);
    hipFree(ds);
    return 0;
}