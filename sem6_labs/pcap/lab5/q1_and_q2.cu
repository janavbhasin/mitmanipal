#include<stdio.h>
#include <hip/hip_runtime.h>

__global__ void vec_add(int*A,int*B,int*C,int n)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if (i<n)
    {
        C[i]=A[i]+B[i];
    }
}
int main(void)
{
    int n;
    printf("enter no of elemnts:");
    scanf("%d",&n);
    int*a=(int*)malloc(n*sizeof(int));
    int*b=(int*)malloc(n*sizeof(int));
    int*c=(int*)malloc(n*sizeof(int));
    int *da,*db,*dc;
    hipMalloc((void**)&da,n*sizeof(int));
    hipMalloc((void**)&db,n*sizeof(int));
    hipMalloc((void**)&dc,n*sizeof(int));
    printf("enter elements of vector 1:\n");
    for(int i=0;i<n;i++)
    {
        scanf("%d",&a[i]);
    }
    printf("enter elements of vector 2:\n");
    for(int i=0;i<n;i++)
    {
        scanf("%d",&b[i]);
    }
    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(db,b,n*sizeof(int),hipMemcpyHostToDevice);
    int bl= 256;
    int num = (n + bl - 1) / bl;
    vec_add<<<num,bl>>>(da,db,dc,n);
    hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("Result: \n");
    for (int i = 0; i < n; i++) 
    {
        printf("%d ", c[i]);
    }
    printf("\n");
    free(a);
    free(b);
    free(c);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}