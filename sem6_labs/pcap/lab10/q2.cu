#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 10
#define K 5
#define BLOCK_SIZE 256
__constant__ int d_kernel[K];
__global__ void conv1D(int *d_input, int *d_output, int n, int k)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int radius = k / 2;

    int sum = 0;
    if (tid < n)
    {
        for (int j = -radius; j <= radius; ++j)
        {
            int idx = tid + j;
            if (idx >= 0 && idx < n)
            {
                sum += d_input[idx] * d_kernel[j + radius];
            }
        }
        d_output[tid] = sum;
    }
}
void initializeData(int *input, int *kernel, int n, int k)
{
    for (int i = 0; i < n; ++i)
    {
        input[i] = rand() % 10;
    }
    for (int i = 0; i < k; ++i)
    {
        kernel[i] = 1;
    }
}
void printArray(int *arr, int n)
{
    for (int i = 0; i < n; ++i)
    {
        printf("%d ", arr[i]);
    }
    printf("\n");
}
int main()
{
    int *h_input, *h_kernel, *h_output;
    int size_input = N * sizeof(int);
    int size_output = N * sizeof(int);
    hipMallocManaged(&h_input, size_input);
    hipMallocManaged(&h_kernel, K * sizeof(int));
    hipMallocManaged(&h_output, size_output);
    initializeData(h_input, h_kernel, N, K);
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, K * sizeof(int));
    int gridSize = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim(gridSize);
    conv1D<<<gridDim, blockDim>>>(h_input, h_output, N, K);
    hipDeviceSynchronize();
    printf("Input Array:\n");
    printArray(h_input, N);
    printf("\nKernel Array:\n");
    printArray(h_kernel, K);
    printf("\nResult of 1D Convolution:\n");
    printArray(h_output, N);
}