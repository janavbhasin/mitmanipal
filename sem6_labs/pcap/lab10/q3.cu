#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 10
#define BLOCK_SIZE 256
__global__ void inclusiveScan(int *d_in, int *d_out, int n)
{
    __shared__ int temp[BLOCK_SIZE];
    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_tid < n)
    {
        temp[tid] = d_in[global_tid];
    }
    else
    {
        temp[tid] = 0;
    }
    __syncthreads();
    for (int offset = 1; offset < blockDim.x; offset *= 2)
    {
        int val = 0;
        if (tid >= offset)
        {
            val = temp[tid - offset];
        }
        __syncthreads();
        temp[tid] += val;
        __syncthreads();
    }
    if (global_tid < n)
    {
        d_out[global_tid] = temp[tid];
    }
}
void initializeArray(int *arr, int n)
{
    for (int i = 0; i < n; ++i)
    {
        arr[i] = 1;
    }
}
void printArray(int *arr, int n)
{
    for (int i = 0; i < n; ++i)
    {
        printf("%d ", arr[i]);
    }
    printf("\n");
}
int main()
{
    int h_in[N], h_out[N];
    int *d_in, *d_out;
    int size = N * sizeof(int);
    initializeArray(h_in, N);
    hipMalloc((void **)&d_in, size);
    hipMalloc((void **)&d_out, size);
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    inclusiveScan<<<gridDim, blockDim>>>(d_in, d_out, N);
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    printf("Input Array:\n");
    printArray(h_in, N);
    printf("\nInclusive Scan Result:\n");
    printArray(h_out, N);
}