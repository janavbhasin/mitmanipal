#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 4
__global__ void matrixMul(int *a, int *b, int *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n)
    {
        int sum = 0;
        for (int k = 0; k < n; ++k)
        {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}
void printMatrix(int *matrix, int n)
{
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            printf("%d ", matrix[i * n + j]);
        }
        printf("\n");
    }
}
int main()
{
    int *a, *b, *c;
    int size = N * N * sizeof(int);
    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);
    for (int i = 0; i < N * N; ++i)
    {
        a[i] = 1;
        b[i] = 2;
    }
    dim3 blockDim(2, 2);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);
    matrixMul<<<gridDim, blockDim>>>(a, b, c, N);
    hipDeviceSynchronize();
    printf("Matrix A:\n");
    printMatrix(a, N);
    printf("\nMatrix B:\n");
    printMatrix(b, N);
    printf("\nResultant Matrix C (A x B):\n");
    printMatrix(c, N);
}