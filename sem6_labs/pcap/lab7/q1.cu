#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
__global__ void word_count(char *str, char *key, int *ind, int *res)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int si = ind[idx], ei = ind[idx + 1], i = 0, i1 = 0, i2 = 0, is_eq = 1;
    char word[100];
    for (i = 0; i < (ei - si - 1); i++)
    {
        word[i] = str[si + 1 + i];
    }
    word[i] = '\0';
    while (word[i1] != '\0' && key[i2] != '\0')
    {
        if (word[i1] == key[i2])
        {
            i1++;
            i2++;
        }
        else
        {
            is_eq = 0;
            break;
        }
    }
    if (is_eq == 1 && key[i2] == '\0' && word[i1] == '\0')
    {
        atomicAdd(res, 1);
    }
}
int main()
{
    char str[100], key[20];
    printf("Enter the string: ");
    scanf(" %[^\n]s", str);
    printf("Enter key: ");
    scanf("%s", key);
    int i = 0, l1 = strlen(str), l2 = strlen(key), count = 1;
    for (i = 0; i < l1; i++)
    {
        if (str[i] == ' ')
        {
            count++;
        }
    }
    int res = 0, wi = 0, *ind = (int *)malloc((count + 1) * sizeof(int));
    ind[0] = -1;
    for (i = 0; i < l1; i++)
    {
        if (str[i] == ' ')
        {
            ind[++wi] = i;
        }
    }
    ind[++wi] = l1;
    char *d_str, *d_key;
    int *d_ind, *d_res;
    hipMalloc((void **)&d_str, l1 * sizeof(char));
    hipMalloc((void **)&d_key, l2 * sizeof(char));
    hipMalloc((void **)&d_ind, (count + 1) * sizeof(int));
    hipMalloc((void **)&d_res, sizeof(int));
    hipMemcpy(d_str, str, l1 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, l2 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_ind, ind, (count + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_res, &res, sizeof(int), hipMemcpyHostToDevice);
    word_count<<<1,count+1>>>(d_str, d_key, d_ind, d_res);
    hipMemcpy(&res, d_res, sizeof(int), hipMemcpyDeviceToHost);
    printf("Total occurrences of '%s' in '%s' is %d\n", key, str,res);
}                           