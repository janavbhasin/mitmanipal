#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
__global__ void kernel(char* sin, int* sin_len, char* sout)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x,si=0;
    for (int i = 0; i < idx; i++)
    {
        si += (*sin_len)-i;
    }
    int total_chars = (*sin_len) - idx;
    for (int i = 0; i < total_chars; i++)
    {
        sout[si++] = sin[i];
    }
}    
int main()
{
    char sin[100], sout[100],* d_sin,* d_sout;
    printf("Enter s: ");
    scanf("%s", sin);
    int sin_len = strlen(sin),*d_sin_len,sout_len = 0;
    for (int i = 0; i < sin_len; i++)
    {
        sout_len += (i+1);
    }
    hipMalloc((void**) &d_sin, sin_len * sizeof(char));
    hipMalloc((void**) &d_sin_len, sizeof(int));
    hipMalloc((void**) &d_sout, (sout_len + 1) * sizeof(char));
    hipMemcpy(d_sin, sin, sin_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_sin_len, &sin_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sout, sout, (sout_len + 1) * sizeof(char), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel <<<1, sin_len>>> (d_sin, d_sin_len, d_sout);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(sout, d_sout, (sout_len + 1) * sizeof(char), hipMemcpyDeviceToHost);
    sout[sout_len] = '\0';
    printf("rs: %s\n", sout);
    printf("Kernel execution time: %f ms\n", milliseconds);
}